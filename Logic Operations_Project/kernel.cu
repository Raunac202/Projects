﻿
#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>

#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

//THIS VERSION OF THE PROGRAM UTILIZES GPU LIBRARIES TO MAKE IT RUN FASTER

//THIS IS A LOGIC OPERATOR PROGRAM
//FIRST TWO SETS ARE CREATED WITH RANDOM INTEGERS, NO DUPLICATE VALUES ARE PRESENT
//THE USER CHOOSES THE SIZE OF BOTH SETS
//EX: IF SET1 SIZE IS 9, THEN SET1 WILL HAVE 9 RANDOM ELEMENTS THAT ARE RANGED FROM -9 TO +9
//AFTER BOTH SETS ARE CREATED, USER CAN PEFORM LOGIC OPERATIONS
//EX: THE INTERSECTION/AND OPERATION OUTPUTS ELEMENTS THAT ARE PRESENT IN BOTH SET1 AND SET2

//THESE ARE THE AVALIABLE OPTIONS THE USER CAN CHOOSE:
// 1. SET1 OR SET2 (Values that belong in SET1 or SET2)
// 2. SET1 AND SET2 (Values that belong in SET1 and SET2)
// 3. SET1 - SET2 (Values that belong in SET1 and not SET2)
// 4. SET2 - SET1 (Values that belong in SET2 and not SET1)
// 5. SET1 SYMMETRIC SET2 (Values that belong in SET1 or SET2, but not their intersection)

//****NOTE: THE SYMMETRIC OPERATION SOMETIMES DOES NOT WORK DUE TO "OUT OF RANGE" ERRORS, 
//HOWEVER, FUTURE IMPLEMENTATIONS WILL FIX THIS ERROR

int rand_custom();
int Size, Size1, Size2;

void ask_usertologic(int[], int, int[], int);
void or_operation(int [], int, int [], int, string);
void and_operation(int[], int, int[], int, string);
void relative_operation(int[], int, int[], int, string);
void symmetric_operation(int[], int, int[], int, string);

void print_elements(string, int[], int);

int main(){
	cout << endl;

	srand(time(0)); //Time function
	
	//Set size for both Sets
	cout << "Please enter the number of elements for the first set: ";
	cin >> Size;
	Size1 = Size;
	thrust::host_vector<int> Set1(Size);
	cout << "Please enter the number of elements for the second set: ";
	cin >> Size;
	Size2 = Size;
	thrust::host_vector<int> Set2(Size);

//	thrust::host_vector<int> Set1(Size);
//	thrust::host_vector<int> Set2(Size);

	//Generate random for both sets
	Size = Size1;
	thrust::generate(Set1.begin(), Set1.end(), rand_custom);
	Size = Size2;
	thrust::generate(Set2.begin(), Set2.end(), rand_custom);

	cout << "SET1 consists of " << Set1.size() << " random numbers generated between -" << Set1.size() << " and +" << Set1.size() << endl;
	cout << "SET2 consists of " << Set2.size() << " random numbers generated between -" << Set2.size() << " and +" << Set2.size() << endl;

	//Transfer to device
	thrust::device_vector<int> d_Set1 = Set1;
	thrust::device_vector<int> d_Set2 = Set2;

	//Sort both sets
	thrust::sort(d_Set1.begin(), d_Set1.end());
	thrust::sort(d_Set2.begin(), d_Set2.end());

	//Transfer back to host
	thrust::copy(d_Set1.begin(), d_Set1.end(), Set1.begin());
	thrust::copy(d_Set2.begin(), d_Set2.end(), Set2.begin());

	//Convert both vectors to arrays
	int* arr1 = &Set1[0];
	int* arr2 = &Set2[0];

	//Print both sets
	print_elements("SET1", arr1, Set1.size());
	print_elements("SET2", arr2, Set2.size());

	//Logical Operations
	string input;
	cout << "Would you like to perform an Operation (yes/no)?: ";
	cin >> input;
	if (input == "yes") {
		ask_usertologic(arr1, Set1.size(), arr2, Set2.size());
		while (input == "yes") {
			cout << "Would you like to perform another Operation (yes/no)?: ";
			cin >> input;
			if (input != "yes") {
				cout << endl;
				break;
			}
			else {
				ask_usertologic(arr1, Set1.size(), arr2, Set2.size());
			}
		}
	}
	else {
		cout << endl;
		cout << "End of Program." << endl;
	}

	cout << "End of Program." << endl;
	cout << endl;
	
	return 0;
}
int rand_custom() {
	return (rand() % (Size * 2 + 1)) - Size;
}
void print_elements(string set, int arr[], int sz) {
	cout << set << ": ";
	for (unsigned int i = 0; i < sz; ++i) {
		if (sz - 1 == i) {
			cout << arr[i];
		}
		else {
			cout << arr[i] << ", ";
		}
	}
	cout << endl;
	cout << endl;
	return;
}
void ask_usertologic(int arr1[], int sz1, int arr2[], int sz2) {
	string input;
	cout << "Which Operation should be performed on SET1 and SET2 (1/2/3/4/5)?" << endl;
	cout << "1. SET1 OR SET2 (Values that belong in SET1 or SET2)" << endl;
	cout << "2. SET1 AND SET2 (Values that belong in SET1 and SET2)" << endl;
	cout << "3. SET1 - SET2 (Values that belong in SET1 and not SET2)" << endl;
	cout << "4. SET2 - SET1 (Values that belong in SET2 and not SET1)" << endl;
	cout << "5. SET1 SYMMETRIC SET2 (Values that belong in SET1 or SET2, but not their intersection)" << endl;
	cout << endl;
	cout << "Operation: ";
	cin >> input;
	cout << endl;
	if (input == "1") {
		or_operation(arr1, sz1, arr2, sz2, "SET1 OR SET2");
	}
	else if (input == "2") {
		and_operation(arr1, sz1, arr2, sz2, "SET1 AND SET2");
	}
	else if (input == "3") {
		relative_operation(arr1, sz1, arr2, sz2, "SET1 - SET2");
	}
	else if (input == "4") {
		relative_operation(arr2, sz2, arr1, sz1, "SET2 - SET1");
	}
	else if (input == "5") {
		symmetric_operation(arr1, sz1, arr2, sz2, "SET1 SYMMETRIC SET2");
	}
	return;
}
void or_operation(int arr1[], int sz1, int arr2[], int sz2, string set) {
	thrust::host_vector<int> Set3;
	thrust::host_vector<int> Set3_cpy;
	for (unsigned int i = 0; i < sz1; ++i) {
		Set3.push_back(arr1[i]);
	}
	for (unsigned int i = 0; i < sz2; ++i) {
		Set3.push_back(arr2[i]);
	}

	int cnt = 0;
	for (unsigned int i = 0; i < Set3.size(); ++i) {
		for (unsigned int j = i + 1; j < Set3.size(); ++j) {
			if (Set3[j] == Set3[i]) {
				cnt++;
			}
			else {

			}
		}
		if (cnt == 0) {
			Set3_cpy.push_back(Set3[i]);
		}
		else {
			cnt = 0;
		}
	}
	//Transfer to device
	thrust::device_vector<int> d_Set3 = Set3_cpy;

	//Sort set
	thrust::sort(d_Set3.begin(), d_Set3.end());

	//Transfer back to host
	thrust::copy(d_Set3.begin(), d_Set3.end(), Set3_cpy.begin());

	//Convert vector to array and print new set
	int* arr3 = &Set3_cpy[0];
	print_elements("SET1 OR SET2", arr3, Set3_cpy.size());
	return;
}
void and_operation(int arr1[], int sz1, int arr2[], int sz2, string set) {
	vector<int> Set3;
	int cnt = 0;
	for (unsigned int i = 0; i < sz1; ++i) {
		for (unsigned int j = 0; j < sz2; ++j) {
			if (arr2[j] == arr1[i]) {
				cnt++;
			}
			else {

			}
		}
		if (cnt > 0) {
			Set3.push_back(arr1[i]);
			cnt = 0;
		}
		else {
			cnt = 0;
		}
	}
	//Convert vector to array and print new set
	int* arr3 = &Set3[0];
	print_elements(set, arr3, Set3.size());
	return;
}
void relative_operation(int arr1[], int sz1, int arr2[], int sz2, string set) {
	vector<int> Set3;
	int cnt = 0;
	for (unsigned int i = 0; i < sz1; ++i) {
		for (unsigned int j = 0; j < sz2; ++j) {
			if (arr2[j] == arr1[i]) {
				cnt++;
			}
			else {

			}
		}
		if (cnt == 0) {
			Set3.push_back(arr1[i]);
		}
		else {
			cnt = 0;
		}
	}
	//Convert vector to array and print new set
	int* arr3 = &Set3[0];
	print_elements(set, arr3, Set3.size());
	return;
}
void symmetric_operation(int arr1[], int sz1, int arr2[], int sz2, string set) {
	thrust::host_vector<int> Set3;
	int cnt = 0;
	int cnt2 = 0;

	while (cnt != sz1) {
		for (unsigned int i = 0; i < sz1; ++i) {
			if (arr1[i] == arr2[cnt]) {
				cnt2++;
			}
			else {

			}
		}
		if (cnt2 == 0) {
			Set3.push_back(arr2[cnt]);
		}
		else {
			cnt2 = 0;
		}
		cnt++;
	}
	cnt = 0;
	cnt2 = 0;
	while (cnt != sz2) {
		for (unsigned int i = 0; i < sz2; ++i) {
			if (arr2[i] == arr1[cnt]) {
				cnt2++;
			}
			else {

			}
		}
		if (cnt2 == 0) {
			Set3.push_back(arr1[cnt]);
		}
		else {
			cnt2 = 0;
		}
		cnt++;
	}

	//Transfer to device
	thrust::device_vector<int> d_Set3 = Set3;

	//Sort both sets
	thrust::sort(d_Set3.begin(), d_Set3.end());

	//Transfer back to host
	thrust::copy(d_Set3.begin(), d_Set3.end(), Set3.begin());

	//Convert vector to array and print new set
	int* arr3 = &Set3[0];
	print_elements("SET1 SYMMETRIC SET2", arr3, Set3.size());
	return;
}
